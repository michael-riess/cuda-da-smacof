#include "hip/hip_runtime.h"
#define _POSIX_C_SOURCE 200809L
#define BILLION 1000000000L

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <unistd.h>
#include <time.h>
#include <stdbool.h>
#include <string.h>
#include <float.h>

#include "distance.h"
#include "guttman.h"
#include "matrix-read.h"
#include "temperature.h"
#include "stress.h"
#include "analysis.h"
#include "random.h"


// Host code
int main(int argc, char** argv) {

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int blocks;     //number of blocks
    int threads;    //number of threads per block
    
    int m;          // number of items / objects; aka 'N'
    int n;          // dimensions of high-dimensional space;
    int s;          // dimension of low-dimensional space; aka 'L'
    double epsilon;    // threshhold for the stress variance; aka 'ε'
    int k_max;      // maximum number of iterations; aka 'MAX'
    bool track_median;

    int iterations = false; // number of test runs for gathering average performance

    float* matrix;

    // validate arguments
    if(argc > 9) {
        fprintf(stderr, "\nToo Many Arguments\n");
        return 1;
    } else if(argc < 8) {
        fprintf(stderr, "\nToo Few Arguments\n");
        return 1;
    } else if (argc == 9) {
        track_median = (strncmp(argv[8], "median", 5) == 0) ? true : false;
    }

    blocks = atoi(argv[2]);
    threads = atoi(argv[3]);
    s = atoi(argv[4]);
    epsilon = strtof(argv[5], NULL);
    k_max = atoi(argv[6]);
    iterations = atoi(argv[7]);

    // read in matrix from file
    readMatrix(argv[1], &matrix, &m, &n);

    //fprintf(stderr, "\nM: %i, N: %i, S: %i\nBlocks: %i, Threads: %i\n", m, n, s, blocks, threads);

    size_t size_D = m*m*sizeof(float);     // total size in memeory of dissimilarity & distance arrays
    size_t size_Y = m*s*sizeof(float);     // total size in memory of low-dimensional array;

    float* Delta = (float*)malloc(size_D);      // pointer to flattened MxM dissimilarity matrix; aka 'Δ' aka 'D'
    float* Delta_prime = (float*)malloc(size_D);// pointer to temperature based dissimilarity matrix; aka '⧊' aka 'delta hat'
    float* Y = (float*)malloc(size_Y);          // MxS set of finding points in the low-dimensional space
    float* D = (float*)malloc(size_D);          // MxM matrix of euclidean distance in target-dimensional space; aka 'projD'

    // Set up environment for tracking median results
    float* Y_med;
    struct stress* Stresses;
    if (track_median) {
        Stresses = (struct stress*)malloc(iterations*sizeof(struct stress));
        Y_med = (float*)malloc(size_Y*iterations);
        for (int i = 0; i < (m*s*iterations); i++) {
            Y_med[i] = 0.0;
        }
    }

    // compute initial dissimiliary matrix
    computeEuclideanDistances(matrix, Delta, m, n, m*n*sizeof(float), size_D, blocks, threads);

    double total_stress = 0.0;
    double max_stress = 0.0;
    double min_stress = DBL_MAX;
    unsigned long total_time = 0;
    struct timespec* timer;


    for(int iter = 0; iter < iterations; iter++) {
        
        timer = startTimer();

        // create initial random solution Y^[0]
        matrixRandomPopulate(Y, m, s, blocks, threads);

        // compute first distance matrix from random Y^[0]
        computeEuclideanDistances(Y, D, m, s, size_Y, size_D, blocks, threads);
        
        int k = 0;           // current interation
        double error = 1.0f;  // error value to determine if close enough approximation in lower dimensional space

        double prev_stress = 0.0f;
        double stress = 0.0f;

        while(k < k_max && error > epsilon) {

            // perform guttman transform
            computeGuttmanTransform(handle, Y, D, Delta, m, s, size_Y, size_D, blocks, threads);
            computeEuclideanDistances(Y, D, m, s, size_Y, size_D, blocks, threads);

            //calculate STRESS
            stress = computeStress(Delta, D, size_D, m, blocks, threads);

            // update error and prev_stress values
            error = fabs(stress - prev_stress);
            prev_stress = stress;
            stress = 0.0f;

            k += 1;
        }

        // end time
        total_time += stopTimer(timer);

        // compute normalized stress for comparing mapping quality
        stress = computeNormalizedStressSerial(Delta, D, m);

        // sum stress values for computing average stress
        total_stress += stress;

        // maintain maximum stress
        if(stress > max_stress) {
            max_stress = stress;
        }

        //maintain minimum stress
        if(stress < min_stress) {
            min_stress = stress;
        }

        // if tracking median results, 
        if (track_median) {
            for (int i = 0; i < (m*s); i++) {
                Y_med[(m*s*iter)+i] = Y[i];
            }
            Stresses[iter].value = stress;
            Stresses[iter].index = iter;
        }
    }

    // print average results after 'iterations' number of test
    printf("\nAVG_TIME: %lf\nAVG_STRESS: %0.8lf\nMAX_STRESS: %0.8lf\nMIN_STRESS: %0.8lf\n", (double)(((long double)total_time/(long double)iterations)/(long double)BILLION), (total_stress/((double)iterations)), max_stress, min_stress);


    // if median is being tracked, print median results including median solution
    if (track_median) {
        struct stress* med = median(Stresses, iterations);
        printf("MEDIAN_STRESS: %0.8lf\nMEDIAN_SOLUTION: [\n", med->value);
        for(int i = 0; i < m; i++) {
            for(int j = 0; j < s; j++) {
                printf("%0.8f", Y_med[(med->index*m*s)+(i*s)+j]);
                if (j != s-1) {
                    printf(" ");
                }
            }
            printf("\n");
        }
        printf("]\n");
        free(Y_med);
        free(Stresses);
    }

    free(matrix);
    free(Delta);
    free(Y);
    free(D);
}