#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#include <stdbool.h>


// cuda macro for ensuring cuda errors are logged
#define __cuda__(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA-Assert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


/* KERNEL: Set up curand environment for populating matrix with pseudorandom values
*/
__global__ void cuda_rand_init(hiprandState *state, unsigned int size, int seed) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x;
        idx < size;
        idx += blockDim.x * gridDim.x
    ){
        hiprand_init(seed, idx, 0, &state[idx]);
     }
}


/* KERNEL: Populate matrix with pseudorandom values
*/
__global__ void cuda_rand(hiprandState *state, float *matrix, unsigned int size) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x;
        idx < size;
        idx += blockDim.x * gridDim.x
    ){
        matrix[idx] = hiprand_uniform(&state[idx]);
    }
}


/* Populate initial solution matrix with pseudorandom values between 0 and 10
*/
void matrixRandomPopulate(float* matrix, int m, int s, int blocks, int threads) {
    float* cuda_matrix;
    hiprandState* cuda_state;
    __cuda__( hipMalloc(&cuda_matrix, m*s*sizeof(float)) );
    __cuda__( hipMalloc(&cuda_state, m*s*sizeof(hiprandState)) );

    // initialize curand state with pseudorandom value for different initial pseudorandom solutions across executions
    srand(time(NULL));
    cuda_rand_init<<<blocks, threads>>>(cuda_state, m*s, (float)rand()/((float)RAND_MAX/10.0f));

    // populate initial solution matrix with pseudorandom values
    cuda_rand<<<blocks, threads>>>(cuda_state, cuda_matrix, m*s);
    __cuda__( hipMemcpy(matrix, cuda_matrix, m*s*sizeof(float), hipMemcpyDeviceToHost) );
    __cuda__( hipFree(cuda_matrix) );
    __cuda__( hipFree(cuda_state) );
}