#define _POSIX_C_SOURCE 200809L

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"


// cuda macro for ensuring cuda errors are logged
#define __cuda__(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA-Assert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


/* KERNEL: Calculate non-diagonal elements of the Guttman Transform Matrix
*/
__global__ void guttmanPart1(float *D, float *projD, float *GT_B, int dataRows){
    
    for (unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
        ix < (dataRows * (dataRows + 1) / 2);
        ix += blockDim.x * gridDim.x
    ){
        // generate 2D indeces from 1D index, ix, in flattened matrix.
        int i = ix / (dataRows + 1);
        int j = ix % (dataRows + 1);
        // if generated indeces lie outside of lower triangle, generate new ones inside it
        if (j > i) {
            i = dataRows - i - 1;
            j = dataRows - j;
        }

        //non-diagonal elements
        if (i != j){
            unsigned int idx = i * dataRows + j;

            if (projD[idx] != 0.0 ){
                GT_B[idx] = -(D[idx]) / projD[idx];
                GT_B[j * dataRows + i] = GT_B[idx];

            }else{
                GT_B[idx] = 0.0f;
                GT_B[j * dataRows + i] = GT_B[idx];
            }
        }
    }
    
}


/* KERNEL: Calculate diagonal elements of the Guttman Transform Matrix
*/
__global__ void guttmanPart2(float *GT_B, int dataRows){

    for (unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
        idx < dataRows;
        idx += blockDim.x * gridDim.x
    ) {

        // diagonal elements
        int i = idx * dataRows + idx;
		GT_B[i] = 0.0f;
        for (int k = 0; k < dataRows; k++){
            if(idx != k) {                          
                GT_B[i] -= GT_B[idx * dataRows + k];
            }
        }
    }
}


/* KERNEL: Generate Moore-Penrose Inverse matrix
*/
__global__ void moorePenroseInverse(float* V, int dataRows) {
    for (unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
        ix < (dataRows * (dataRows + 1) / 2);
        ix += blockDim.x * gridDim.x
    ) {
        float N = (float)dataRows;

        // generate 2D indeces from 1D index, ix, in flattened matrix.
        int i = ix / (dataRows + 1);
        int j = ix % (dataRows + 1);
        // if generated indeces lie outside of lower triangle, generate new ones inside it
        if (j > i) {
            i = dataRows - i - 1;
            j = dataRows - j;
        }

        if (j != i) {
            V[j * dataRows + i] = -1.0f/(N*N);
            V[i * dataRows + j] = V[j * dataRows + i];
        } else {
            V[j * dataRows + i] = (N-1.0f)/(N*N);
        }
    }
}


/* Compute the Guttman Transform with gpu
*  Y` <= V` * B * Y
*/
void computeGuttmanTransform(hipblasHandle_t handle, float* Y, float* D, float* Delta, int m, int s, size_t size_Y, size_t size_D, int blocks, int threads) {
    float* cuda_D;
    float* cuda_Delta;
    float* cuda_B;
    float* cuda_B_prime;
    float* cuda_V;
    float* cuda_Y;
    float* cuda_Y_prime;

    // necessary scalar factors for cublasSgemm function
    float alpha = 1.0f;
    float beta = 0.0f;

    __cuda__( hipMalloc(&cuda_D, size_D) );
    __cuda__( hipMalloc(&cuda_B, size_D) );
    __cuda__( hipMalloc(&cuda_Delta, size_D) );

    __cuda__( hipMemcpy(cuda_D, D, size_D, hipMemcpyHostToDevice) );
    __cuda__( hipMemcpy(cuda_Delta, Delta, size_D, hipMemcpyHostToDevice) );

    // generate the guttman transform matrix, B
    guttmanPart1<<<blocks, threads>>>(cuda_Delta, cuda_D, cuda_B, m);
    __cuda__( hipPeekAtLastError() );
    __cuda__( hipFree(cuda_D) );
    __cuda__( hipFree(cuda_Delta) );
    guttmanPart2<<<blocks, threads>>>(cuda_B, m);
    __cuda__( hipPeekAtLastError() );

    __cuda__( hipMalloc(&cuda_V, size_D) );
    __cuda__( hipMalloc(&cuda_B_prime, size_D) );

    // generate the moore-penrose inverse matrix, V`
    moorePenroseInverse<<<blocks, threads>>>(cuda_V, m);
    __cuda__( hipPeekAtLastError() );


    /**** cuda_Y_prime <= cuda_V * cuda_B * cuda_Y ****
    * --------------------------------------------------
    * **** NOTE: **** 
    * Cublas expects column-major-order matrices, therefore row-major-ordered matrices (standard C) are viewed as
    * transposes. Additinally, we know that (A*B)^T == B^T * A^T. Thus we simply switch the order of matrices A
    * and B in the cublassgemm call and recieve the proper C value.
    * **** Cublas documention : ****
    * http://developer.download.nvidia.com/compute/cuda/2_0/docs/CUBLAS_Library_2.0.pdf
    * http://rpm.pbone.net/index.php3/stat/45/idpl/12463013/numer/3/nazwa/cublasSgemm
    * **** Explenation: ****
    * https://www.christophlassner.de/using-blas-from-c-with-row-major-data.html
    */

    // cuda_B_prime <= cuda_V * cuda_B   //   B` <= V` * B
    // multiply cuda_V and cuda_B e.g. the moore-penrose inverse matrix and the guttman transform matrix.
    hipblasSgemm (handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, m, &alpha, cuda_B, m, cuda_V, m, &beta, cuda_B_prime, m);

    __cuda__( hipFree(cuda_B) );
    __cuda__( hipFree(cuda_V) );

    __cuda__( hipMalloc(&cuda_Y, size_Y) );
    __cuda__( hipMalloc(&cuda_Y_prime, size_Y) );

    __cuda__( hipMemcpy(cuda_Y, Y, size_Y, hipMemcpyHostToDevice) );

    // cuda_Y_prime <= cuda_B_prime * cuda_Y    //   Y` <= B` * Y
    // multply cuda_B_prime and cuda_Y e.g. the result of the previouse matrix multiplication and the latest solution matrix
    hipblasSgemm (handle, HIPBLAS_OP_N, HIPBLAS_OP_N, s, m, m, &alpha, cuda_Y, s, cuda_B_prime, m, &beta, cuda_Y_prime, s);

    // save new cuda_Y_prime matrix to Y
    __cuda__( hipMemcpy(Y, cuda_Y_prime, size_Y, hipMemcpyDeviceToHost) );

    __cuda__( hipFree(cuda_B_prime) );
    __cuda__( hipFree(cuda_Y) );
    __cuda__( hipFree(cuda_Y_prime) );
}



void matrixMultiplySerial(float* A, float* B, float* C, int a_m, int a_n, int b_m, int b_n) {
    if (a_n != b_m) {
        return;
    }
    for (int i = 0; i < a_m; i++) {
        for (int j = 0; j < b_n; j++) {
            float sum = 0.0f;
            for (int k = 0; k < b_m; k++) {
                sum = sum + A[(i * a_n) + k] * B[(k * b_n) + j];
            }
            C[(i * b_n) + j] = sum;
        }
    }
}


/* Compute the Guttman Transform without gpu
*  Y` <= V` * B * Y
*/
void computeGuttmanTransformSerial(float** Y, float* D, float* Delta, int m, int s, size_t size_Y, size_t size_D) {
    float* B = (float*)malloc(size_D);
    float* V = (float*)malloc(size_D);
    float* C = (float*)malloc(size_D);
    float* Y_prime;

    // generate guttman transform matrix
    // calculate non-diagonal values
    for (int i = 0; i < m; i++) {
        for (int j = i; j < m; j++) {
            int ix = (i * m) + j;
            if (i == j || D[ix] == 0.0f) {
                B[ix] = 0.0f;
            } else {
                B[ix] = -(Delta[ix]) / D[ix];
            }
            B[(j * m) + i] = B[ix];
        }
    }

    // calculate diagonal values
    for (int i = 0; i < m; i++) {
        for (int k = 0; k < m; k++){
            if(i != k) {                          
                B[(i * m) + i] -= B[(i * m) + k];
            }
        }
    }

    // generate moore-penrose inverse matrix
    float N = (float)m;
    for (int i = 0; i < m; i ++) {
        for (int j = i; j < m; j++){
            if (j != i) {
                V[j * m + i] = -1.0f/(N*N);
                V[i * m + j] = V[j * m + i];
            } else {
                V[j * m + i] = (N-1.0f)/(N*N);
            }
        }
    }

    // perform guttman transform
    matrixMultiplySerial(V, B, C, m, m, m, m);
    free(B);
    free(V);
    Y_prime = (float*)malloc(size_Y);
    matrixMultiplySerial(C, *Y, Y_prime, m, m, m, s);
    free(C);
    C = *Y;
    *Y = Y_prime;
    free(C);
}
